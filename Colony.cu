#include "hip/hip_runtime.h"
﻿#include <hip/hip_runtime_api.h> //ToDo: Might be unnecessary include

#include "Colony.cuh"

__device__ void Colony::Init(Ant *Ants, ColonyState *State, AntState *AntStates, Colony *OtherColony)
{
	mArmy = Ants;
	mOtherColony = OtherColony;
	mState = State;
	mArmyStates = AntStates;
	mAge = 0;
	mState->mPoints = 0;
	mAlmostPoints = 0;
	mPointTime = 0;
	mIsWinner = false;
}

__device__ void Colony::AddPoint(int Type, int Time)
{
	if (Type == -1)
	{
		atomicAdd(&mAlmostPoints, -1);
	}
	else if (Type == 0)
	{
		atomicAdd(&mAlmostPoints, 1);
		//if (mPoints == 0)
			//atomicExch(&mPointTime, Time);
	}
	else
	{
		atomicAdd(&mState->mPoints, 1);
		atomicAdd(&mAlmostPoints, -1);
		atomicExch(&mPointTime, Time);
	}
	SetIsWinner();
}

__device__ void Colony::Reset(Ant *Ants, ColonyState *State, AntState *AntStates, Colony *OtherColony)
{
	mArmy = Ants;
	mOtherColony = OtherColony;
	mState = State;
	mArmyStates = AntStates;
	++mAge;
	mState->mPoints = 0;
	mAlmostPoints = 0;
	mPointTime = 0;
	mIsWinner = false;
}

__device__ void Colony::SetIsWinner()
{
	//mPointDiff = mState->mPoints - mOtherColony->mState->mPoints;
	//mOtherColony->mPointDiff = -mPointDiff;

	if (mState->mPoints != mOtherColony->mState->mPoints)
	{
		mIsWinner = mState->mPoints > mOtherColony->mState->mPoints;
		mOtherColony->mIsWinner = !mIsWinner;
	}
	else if (mState->mPoints && (mPointTime != mOtherColony->mPointTime))
	{
		mIsWinner = mPointTime < mOtherColony->mPointTime;
		mOtherColony->mIsWinner = !mIsWinner;
	}
	else if (mAlmostPoints != mOtherColony->mAlmostPoints)
	{
		mIsWinner = mAlmostPoints > mOtherColony->mAlmostPoints;
		mOtherColony->mIsWinner = !mIsWinner;
	}
	//ToDo: might cause overfitting, needs testing
	else if (mAlmostPoints)
	{
		mIsWinner = mPointTime < mOtherColony->mPointTime;
		mOtherColony->mIsWinner = !mIsWinner;
	}
	else
		mIsWinner = mOtherColony->mIsWinner = false;
}

__host__ __device__ bool Colony::operator>(const Colony&other) const
{
	if (mIsWinner != other.mIsWinner)
		return mIsWinner;
	int diff1 = mState->mPoints - mOtherColony->mState->mPoints;
	int diff2 = other.mState->mPoints - other.mOtherColony->mState->mPoints;
	if (diff1 != diff2)
		return diff1 > diff2;
	if (mState->mPoints != other.mState->mPoints)
		return mState->mPoints > other.mState->mPoints;
	if (mAlmostPoints != other.mAlmostPoints)
		return mAlmostPoints > other.mAlmostPoints;
	
	//Calculate ratio

	//ToDo: Rewrite based on current model

	/*if (mIsWinner != other.mIsWinner)
		return mIsWinner;
	if (mState->mPoints != other.mState->mPoints && !(mState->mPoints * other.mState->mPoints))
		return mState->mPoints;
	if (mState->mPoints != other.mState->mPoints)
		return mState->mPoints > other.mState->mPoints;
	//if (mPoints && (mPointTime != mOtherColony->mPointTime))
	//	return mPointTime < other.mPointTime;
	if (true) //sortwithtrue
	{
		if (mAlmostPoints != other.mAlmostPoints)
			return mAlmostPoints > other.mAlmostPoints;
	}*/
	return false;
}